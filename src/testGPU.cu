#include <iostream>
#include <hip/hip_runtime.h>

// QUICK TEST TO SEE IF CUDAMALLOC AND CUDAMEMCPY WORK

int main()
{
  double* x;
  x=(double*)malloc(sizeof(double));
  x[0]=2.1;
  double* dev_x;

  hipError_t err=hipMalloc(&dev_x, sizeof(double));

  std::cout<<hipGetErrorName(err)<<std::endl;

  err=hipMemcpy(dev_x, x, sizeof(double), hipMemcpyHostToDevice);

  std::cout<<hipGetErrorName(err)<<std::endl;

  double* y;
  y=(double*) malloc(sizeof(double));
  err=hipMemcpy(y, dev_x, sizeof(double), hipMemcpyDeviceToHost);

  std::cout<<hipGetErrorName(err)<<std::endl;

  std::cout<<y<<std::endl;

  return 0;
  
}
