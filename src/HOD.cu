#include "hip/hip_runtime.h"
#include "HOD.h"
#include "Params.h"
#include <iostream>

#if GPU
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#endif

g3lhalo::HOD::HOD(Params *params_)
{
    params = params_;

#if GPU
    param_arr[0] = params->f;
    param_arr[1] = params->alpha;
    param_arr[2] = params->mmin;
    param_arr[3] = params->sigma;
    param_arr[4] = params->mprime;
    param_arr[5] = params->beta;

    CUDA_SAFE_CALL(hipMalloc(&dev_params, 6 * sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpy(dev_params, param_arr, 6 * sizeof(double), hipMemcpyHostToDevice));

#endif
}

g3lhalo::HOD::~HOD()
{
#if GPU
    hipFree(dev_params);
#endif
}

__host__ __device__ double g3lhalo::HOD::Nsat(double m, double *d_params)
{
#if __CUDA_ARCH__
    if (d_params == NULL)
    {
        printf("Problem in g3lhalo::HOD::Nsat\n");
        printf("Returning Nsat=0, but check this!\n");
        return 0;
    }

    double mth = d_params[2];
    double sigma = d_params[3];
    double mprime = d_params[4];
    double beta = d_params[5];
#else
    double mth = params->mmin;
    double sigma = params->sigma;
    double mprime = params->mprime;
    double beta = params->beta;
#endif
    return 0.5 * (1 + erf(log(m / mth) / sigma / 1.414213562)) * pow(m / mprime, beta);
}

__host__ __device__ double g3lhalo::HOD::Ncen(double m, double *d_params)
{
#if __CUDA_ARCH__
    if (d_params == NULL)
    {
        printf("Problem in g3lhalo::HOD::Ncen\n");
        printf("Returning Ncen=0, but check this!\n");
        return 0;
    }
    double alpha = d_params[1];
    double mth = d_params[2];
    double sigma = d_params[3]; 
#else
    double alpha = params->alpha;
    double mth = params->mmin;
    double sigma = params->sigma;
#endif
    return 0.5 * alpha * (1 + erf(log(m / mth) / sigma / 1.414213562));
}

__host__ __device__ double g3lhalo::NsatNsat(double m, HOD *hod1, HOD *hod2, double* d_params1, double* d_params2,
 double A, double epsilon, double scale1, double scale2)
{
    bool sameType = false;
    if (hod1 == hod2)
        sameType = true;

    double Ns1 = hod1->Nsat(m, d_params1);
    if (sameType)
        return (Ns1 + scale1 * scale1 - 1) * Ns1;

    double Ns2 = hod2->Nsat(m, d_params2);
    double result = Ns1 * Ns2 + A * pow(m, epsilon) * sqrt(Ns1 * Ns2);
    if (result < 0)
        return 0;
    return result;
}